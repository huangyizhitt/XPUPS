#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "mfworker.h"

namespace MF{

void MFWorker::PrepareGPUResources()
{
	size_t size_p = m * k;
	size_t size_q = n * k;

#ifdef CAL_PORTION_RMSE
	hipMalloc(&feature, (size_p + size_q + 1) * sizeof(float));
	hipMalloc(&gpu_loss, workers * sizeof(float) * 32);
#else
	hipMalloc(&feature, (size_p + size_q) * sizeof(float));
#endif
	p = feature;
	q = feature + size_p;

	hipMalloc(&gpuR, size * sizeof(MatrixNode));
}

void MFWorker::ReleaseGPUResources()
{
	hipFree(feature);
	hipFree(gpuR);

#ifdef CAL_PORTION_RMSE
	hipFree(gpu_loss);
#endif
}

void MFWorker::PullGPUData()
{
	MatrixNode *cpuR = dm.data.r_matrix.data();
	hipMemcpy(gpuR, cpuR, sizeof(MatrixNode) * size, hipMemcpyHostToDevice);
}


}
