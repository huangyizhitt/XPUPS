#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "mfworker.h"

namespace MF{

void MFWorker::PrepareGPUResources()
{
	size_t size_p = m * k;
	size_t size_q = n * k;

#ifdef CAL_PORTION_RMSE
	loss_size = 32 * workers;
	hipMalloc(&feature, (size_p + size_q + 1) * sizeof(float));

	if(trans_mode != HALFQ_SHM_ACOPY) {
		hipHostMalloc(&loss, sizeof(float) * loss_size);
		hipMalloc(&gpu_loss, sizeof(float) * loss_size);
	} else {
		hipHostMalloc(&loss, sizeof(float) * loss_size * xpu->num_streams);
                hipMalloc(&gpu_loss, sizeof(float) * loss_size * xpu->num_streams);
	}
#else
	hipMalloc(&feature, (size_p + size_q) * sizeof(float));
#endif
	p = feature;
	q = feature + size_p;

	hipMalloc(&gpuR, size * sizeof(MatrixNode));
}

void MFWorker::ReleaseGPUResources()
{
	hipFree(feature);
	hipFree(gpuR);

#ifdef CAL_PORTION_RMSE
	hipFree(gpu_loss);
	hipHostFree(loss);
#endif
}

void MFWorker::PullGPUData()
{
	MatrixNode *cpuR = dm.data.r_matrix.data();
	hipMemcpy(gpuR, cpuR, sizeof(MatrixNode) * size, hipMemcpyHostToDevice);
}

void MFWorker::PinnedBuf(void* buf, size_t size)
{
        hipHostRegister(buf, size, hipHostRegisterDefault);
}

void MFWorker::UnpinnedBuf(void *buf)
{
        hipHostUnregister(buf);
}
}
