#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "mfworker.h"

namespace MF{

void MFWorker::PrepareGPUResources()
{
	size_t size_p = m * k;
	size_t size_q = n * k;

#ifdef CAL_PORTION_RMSE
	hipMalloc(&feature, (size_p + size_q + 1) * sizeof(float));
	hipMalloc(&gpu_loss, size * sizeof(float));
#else
	hipMalloc(&feature, (size_p + size_q) * sizeof(float));
#endif
	p = feature;
	q = feature + size_p;

	hipMalloc(&gpuR, size * sizeof(MatrixNode));
}

void MFWorker::ReleaseGPUResources()
{
	hipFree(feature);
	hipFree(gpuR);

#ifdef CAL_PORTION_RMSE
	hipFree(gpu_loss);
#endif
}

void MFWorker::PullGPUData()
{
	MatrixNode *cpuR = dm.data.r_matrix.data();
	hipMemcpy(gpuR, cpuR, sizeof(MatrixNode) * size, hipMemcpyHostToDevice);
}


void MFWorker::sgd_update_k128_gpu()
{
	hiprandState *rand_state;
    hipMalloc(&rand_state, sizeof(hiprandState)*core_num);
	gpuErr(hipPeekAtLastError());

	init_rand_state<<<((core_num+255)/256),256>>>(rand_state,core_num);
	gpuErr(hipPeekAtLastError());

	int update_vector_size = 128;
	int update_count = (ceil)(1.0 * size / (core_num*update_vector_size));

#ifdef CAL_PORTION_RMSE
	sgd_k128_kernel_hogwild_warp32_loss<<<core_num/4, 128>>>(gpuR, start, size, rand_state, gpu_loss, 
						p, q, k, update_count, update_vector_size, lrate, lambda_p, lambda_q);
	hipMemcpy(loss, gpu_loss, size * sizeof(float), hipMemcpyDeviceToHost);
#else
	sgd_k128_kernel_hogwild_warp32<<<core_num/4, 128>>>(gpuR, start, size, rand_state,
						p, q, k, update_count, update_vector_size, lrate, lambda_p, lambda_q);
#endif

	gpuErr(hipPeekAtLastError());
	hipDeviceSynchronize();
	hipFree(rand_state);
}

}
