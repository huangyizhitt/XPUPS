#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/ipc.h>
#include <sys/shm.h>
#include "mfserver.h"

namespace MF{

void MFServer::PinnedBuf(void* buf, size_t size)
{
	hipHostRegister(buf, size, hipHostRegisterDefault);
}

void MFServer::UnpinnedBuf(void *buf)
{
	hipHostUnregister(buf);
}
}
